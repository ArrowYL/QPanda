#include "hip/hip_runtime.h"
/******************************************************************************
Copyright (c) 2017-2018 Origin Quantum Computing Co., Ltd.. All Rights Reserved.


 
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at 

 	http://www.apache.org/licenses/LICENSE-2.0 
 
Unless required by applicable law or agreed to in writing, software

distributed under the License is distributed on an "AS IS" BASIS,

WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.

See the License for the specific language
governing permissions and 
limitations under the License.

Author:Xue Cheng
Date:2017-12-13
Description: Definition of Cuda function of gates
************************************************************************/
#include "GPUGatesDecl.h"
#include <vector>
#include <algorithm>
#include <cuda_device_runtime_api.h>
#include <>
#include <hip/hip_runtime.h>
using namespace std;
#define QSIZE   size_t

#define SQ2 0.707106781186548
#define PI 3.141592653589793
#define THREADDIM 1024
namespace GATEGPU
{
    //typedef quantumstate QState;

    typedef std::vector<STATE_T> vecdou;
    //typedef std::vector<probability> vecprob;
    __global__ void unitary(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, STATE_T *matr, QSIZE Block);
    __global__ void initState(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim);
    __global__ void Hadamard(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE Block);
    __global__ void Hadamardnew(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE Block);
    __global__ void Hadamarddagger(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE Block);
    __global__ void RX(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE Block, STATE_T costheta, STATE_T sintheta);
    __global__ void RXdagger(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE Block, STATE_T costheta, STATE_T sintheta);
    __global__ void RY(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE Block, STATE_T costheta, STATE_T sintheta);
    __global__ void RYdagger(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE Block, STATE_T costheta, STATE_T sintheta);
    __global__ void RZ(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE Block, STATE_T costheta, STATE_T sintheta);
    __global__ void RZdagger(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE Block, STATE_T costheta, STATE_T sintheta);
    __global__ void NOT(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE Block);
    __global__ void NOTdagger(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE Block);
    __global__ void CNOT(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE Block1, QSIZE Block2);
    __global__ void CNOTdagger(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE Block1, QSIZE Block2);
    __global__ void CR(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE Block1, QSIZE Block2, STATE_T costheta, STATE_T sintheta);
    __global__ void CRdagger(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE Block1, QSIZE Block2, STATE_T costheta, STATE_T sintheta);
    __global__ void iSWAP(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE Block1, QSIZE Block2);
    __global__ void iSWAPdagger(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE Block1, QSIZE Block2);
    __global__ void sqiSWAP(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE Block1, QSIZE Block2);
    __global__ void sqiSWAPdagger(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE Block1, QSIZE Block2);
    __global__ void qubitprob(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE Block, STATE_T *pr);
    __global__ void probsum(STATE_T * pr, STATE_T * prob);
    __global__ void qubitcollapse0(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE Block, STATE_T coef);
    __global__ void qubitcollapse1(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE Block, STATE_T coef);
    __global__ void controlHadamard(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE * block1, QSIZE  block2, QSIZE m);
    __global__ void controlHadamarddagger(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE * block1, QSIZE  block2, QSIZE m);
    __global__ void controlRX(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE * block1, QSIZE  block2, QSIZE m, STATE_T costheta, STATE_T sintheta);
    __global__ void controlRXdagger(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE * block1, QSIZE  block2, QSIZE m, STATE_T costheta, STATE_T sintheta);
    __global__ void controlRY(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE * block1, QSIZE  block2, QSIZE m, STATE_T costheta, STATE_T sintheta);
    __global__ void controlRYdagger(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE * block1, QSIZE  block2, QSIZE m, STATE_T costheta, STATE_T sintheta);
    __global__ void controlRZ(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE * block1, QSIZE  block2, QSIZE m, STATE_T costheta, STATE_T sintheta);
    __global__ void controlRZdagger(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE * block1, QSIZE  block2, QSIZE m, STATE_T costheta, STATE_T sintheta);
    __global__ void toffoli(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE  block0, QSIZE  block1, QSIZE block2);
    __global__ void toffolidagger(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE  block0, QSIZE  block1, QSIZE block2);
    __global__ void multiprob(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, STATE_T * pro, QSIZE * block, QSIZE m, QSIZE dec);
    __global__ void probsumnew1(STATE_T * psireal, STATE_T * psiimag, STATE_T *probtemp, size_t num1, size_t m, size_t Dim, size_t * block);

    //void Hadamard(QState& psi, size_t qn, double error_rate = 0);
    DLLAPI int devicecount();
    DLLAPI bool initstate(QState& psi, QState& psigpu, int);
    DLLAPI bool destroyState(QState& psi, QState& psigpu, size_t sQnum);
    DLLAPI bool clearState(QState& psi, QState& psigpu, size_t sQnum);
    DLLAPI bool Hadamard(QState& psi, size_t qn, double error_rate = 0);
    DLLAPI bool Hadamardnew(QState& psi, size_t qn, double error_rate = 0);
    DLLAPI bool Hadamarddagger(QState& psi, size_t qn, double error_rate = 0);
    DLLAPI bool RX(QState& psi, size_t, double, double error_rate = 0);
    DLLAPI bool RXdagger(QState& psi, size_t, double, double error_rate = 0);
    DLLAPI bool RY(QState& psi, size_t, double, double error_rate = 0);
    DLLAPI bool RYdagger(QState& psi, size_t, double, double error_rate = 0);
    DLLAPI bool RZ(QState& psi, size_t, double, double error_rate = 0);
    DLLAPI bool RZdagger(QState& psi, size_t, double, double error_rate = 0);
    DLLAPI bool NOT(QState& psi, size_t qn, double error_rate = 0);
    DLLAPI bool NOTdagger(QState& psi, size_t qn, double error_rate = 0);
    DLLAPI bool CNOT(QState& psi, size_t, size_t, double error_rate = 0);
    DLLAPI bool CNOTdagger(QState& psi, size_t, size_t, double error_rate = 0);
    DLLAPI bool CR(QState& psi, size_t, size_t, double, double error_rate = 0);
    DLLAPI bool CRdagger(QState& psi, size_t, size_t, double, double error_rate = 0);
    DLLAPI bool iSWAP(QState& psi, size_t, size_t, double error_rate = 0);
    DLLAPI bool iSWAPdagger(QState& psi, size_t, size_t, double error_rate = 0);
    DLLAPI bool sqiSWAP(QState& psi, size_t, size_t, double error_rate = 0);
    DLLAPI bool sqiSWAPdagger(QState& psi, size_t, size_t, double error_rate = 0);
    DLLAPI int qubitmeasure(QState& psi, QSIZE Block);
    DLLAPI bool controlHadamard(QState& psi, Qnum&, double error_rate = 0);
    DLLAPI bool controlHadamarddagger(QState& psi, Qnum&, double error_rate = 0);
    DLLAPI bool controlRX(QState& psi, Qnum&, double, double error_rate = 0);
    DLLAPI bool controlRXdagger(QState& psi, Qnum&, double, double error_rate = 0);
    DLLAPI bool controlRY(QState& psi, Qnum&, double, double error_rate = 0);
    DLLAPI bool controlRYdagger(QState& psi, Qnum&, double, double error_rate = 0);
    DLLAPI bool controlRZ(QState& psi, Qnum&, double, double error_rate = 0);
    DLLAPI bool controlRZdagger(QState& psi, Qnum&, double, double error_rate = 0);
    DLLAPI bool toffoli(QState& psi, size_t, size_t, size_t, double error_rate = 0);
    DLLAPI bool toffolidagger(QState& psi, size_t, size_t, size_t, double error_rate = 0);
    DLLAPI bool qbReset(QState& psi, size_t, double error_rate = 0);
    DLLAPI bool pMeasure(QState&, vecprob&, QSIZE *block, QSIZE m);
    DLLAPI bool pMeasurenew(QState&, vector<pair<size_t, double>>&, Qnum&);
	DLLAPI bool getState(QState &psi,QState &psigpu,int qnum);
    double randGenerator();
}

/***************************************************************************************
Probdis pMeasure(QState&, Qnum&);
********************************************************************************************/


__global__ void GATEGPU:: unitary(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, STATE_T *matr, QSIZE Block)
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;           //thread number
    QSIZE BlockNum = idx / Block;
    QSIZE BlockInt = idx% Block;
    QSIZE realIdx = BlockNum * 2 * Block + BlockInt;
    QSIZE corIdx = realIdx + Block;

    if (corIdx < Dim)
    {
        STATE_T X1 = psireal[realIdx];
        STATE_T X2 = psireal[corIdx];
        STATE_T Y1 = psiimag[realIdx];
        STATE_T Y2 = psiimag[corIdx];
        psireal[realIdx] = X1*matr[0] - Y1*matr[1] + X2*matr[2] - Y2*matr[3];
        psireal[corIdx] = X1*matr[4] - Y1*matr[5] + X2*matr[6] - Y2*matr[7];
        psiimag[realIdx] = X1*matr[1] + Y1*matr[0] + X2*matr[3] + Y2*matr[2];
        psiimag[corIdx] = X1*matr[5] + Y1*matr[4] + X2*matr[7] + Y2*matr[6];
    }

}//checked

__global__ void GATEGPU:: initState(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim)
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;           //thread number

    if (idx < Dim / 2 && idx != 0)
    {
        psireal[idx] = 0;
        psiimag[idx] = 0;
        psireal[idx + Dim / 2] = 0;
        psiimag[idx + Dim / 2] = 0;
    }
    if (0 == idx)
    {
        psireal[0] = 1;
        psiimag[0] = 0;
        psireal[Dim / 2] = 0;
        psiimag[Dim / 2] = 0;
    }
}//checked

__global__ void GATEGPU:: Hadamardnew(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE Block)
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;           //thread number
    //QSIZE BlockNum = idx / Block;
    //QSIZE BlockInt = idx% Block;
    //QSIZE realIdx = BlockNum * 2 * Block + BlockInt;
   // QSIZE corIdx = realIdx + Block;
    QSIZE realIdx;
    QSIZE corIdx;
    STATE_T X1, X2, Y1, Y2;
     for (int i = idx; i < Dim; i += gridDim.x*blockDim.x)
    {
         realIdx = i / (Block<<1) * 2 * Block + i%Block ;
         corIdx = realIdx + Block;
          X1 = psireal[realIdx];
          X2 = psireal[corIdx];
          Y1 = psiimag[realIdx];
          Y2 = psiimag[corIdx];
         psireal[realIdx] = (X1 + X2)*SQ2;
         psireal[corIdx] = (X1 - X2)*SQ2;
         psiimag[realIdx] = (Y1 + Y2)*SQ2;
         psiimag[corIdx] = (Y1 - Y2)*SQ2;

     }
     /*
    if (corIdx < Dim)
    {
        STATE_T X1 = psireal[realIdx];
        STATE_T X2 = psireal[corIdx];
        STATE_T Y1 = psiimag[realIdx];
        STATE_T Y2 = psiimag[corIdx];
        psireal[realIdx] = (X1 + X2)*SQ2;
        psireal[corIdx] = (X1 - X2)*SQ2;
        psiimag[realIdx] = (Y1 + Y2)*SQ2;
        psiimag[corIdx] = (Y1 - Y2)*SQ2;
    }
    */
}//checked
__global__ void GATEGPU::Hadamard(STATE_T * psireal, STATE_T * psiimag, QSIZE Dim, QSIZE Block)
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;           //thread number
    QSIZE BlockNum = idx / Block;
    QSIZE BlockInt = idx% Block;
    QSIZE realIdx = BlockNum * 2 * Block + BlockInt;
    QSIZE corIdx = realIdx + Block;

    if (corIdx < Dim)
    {
        STATE_T X1 = psireal[realIdx];
        STATE_T X2 = psireal[corIdx];
        STATE_T Y1 = psiimag[realIdx];
        STATE_T Y2 = psiimag[corIdx];
        psireal[realIdx] = (X1 + X2)*SQ2;
        psireal[corIdx] = (X1 - X2)*SQ2;
        psiimag[realIdx] = (Y1 + Y2)*SQ2;
        psiimag[corIdx] = (Y1 - Y2)*SQ2;
    }
}//checked

__global__ void GATEGPU:: Hadamarddagger(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE Block)
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;           //thread number
    QSIZE BlockNum = idx / Block;
    QSIZE BlockInt = idx% Block;
    QSIZE realIdx = BlockNum * 2 * Block + BlockInt;
    QSIZE corIdx = realIdx + Block;

    if (corIdx < Dim)
    {
        STATE_T X1 = psireal[realIdx];
        STATE_T X2 = psireal[corIdx];
        STATE_T Y1 = psiimag[realIdx];
        STATE_T Y2 = psiimag[corIdx];
        psireal[realIdx] = (X1 + X2)*SQ2;
        psireal[corIdx] = (X1 - X2)*SQ2;
        psiimag[realIdx] = (Y1 + Y2)*SQ2;
        psiimag[corIdx] = (Y1 - Y2)*SQ2;
    }
}//checked

__global__ void GATEGPU:: RX(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE Block, STATE_T costheta, STATE_T sintheta)
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;           //thread number
    QSIZE BlockNum = idx / Block;
    QSIZE BlockInt = idx% Block;
    QSIZE realIdx = BlockNum * 2 * Block + BlockInt;
    QSIZE corIdx = realIdx + Block;
    if (corIdx < Dim)
    {
        STATE_T X1 = psireal[realIdx];
        STATE_T X2 = psireal[corIdx];
        STATE_T Y1 = psiimag[realIdx];
        STATE_T Y2 = psiimag[corIdx];
        psireal[realIdx] = X1*costheta + Y2*sintheta;
        psireal[corIdx] = Y1*sintheta + X2*costheta;
        psiimag[realIdx] = Y1*costheta - X2*sintheta;
        psiimag[corIdx] = Y2*costheta - X1*sintheta;
    }
}//checked

__global__ void GATEGPU:: RXdagger(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE Block, STATE_T costheta, STATE_T sintheta)
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;           //thread number
    QSIZE BlockNum = idx / Block;
    QSIZE BlockInt = idx% Block;
    QSIZE realIdx = BlockNum * 2 * Block + BlockInt;
    QSIZE corIdx = realIdx + Block;
    if (corIdx < Dim)
    {
        STATE_T X1 = psireal[realIdx];
        STATE_T X2 = psireal[corIdx];
        STATE_T Y1 = psiimag[realIdx];
        STATE_T Y2 = psiimag[corIdx];
        psireal[realIdx] = X1*costheta - Y2*sintheta;
        psireal[corIdx] = -Y1*sintheta + X2*costheta;
        psiimag[realIdx] = Y1*costheta + X2*sintheta;
        psiimag[corIdx] = Y2*costheta + X1*sintheta;
    }
}//checked

__global__ void GATEGPU:: RY(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE Block, STATE_T costheta, STATE_T sintheta)
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;           //thread number
    QSIZE BlockNum = idx / Block;
    QSIZE BlockInt = idx% Block;
    QSIZE realIdx = BlockNum * 2 * Block + BlockInt;
    QSIZE corIdx = realIdx + Block;
    if (corIdx < Dim)
    {
        STATE_T X1 = psireal[realIdx];
        STATE_T X2 = psireal[corIdx];
        STATE_T Y1 = psiimag[realIdx];
        STATE_T Y2 = psiimag[corIdx];
        psireal[realIdx] = X1*costheta - X2*sintheta;
        psireal[corIdx] = X1*sintheta + X2*costheta;
        psiimag[realIdx] = Y1*costheta - Y2*sintheta;
        psiimag[corIdx] = Y2*costheta + Y1*sintheta;
    }
} //checked

__global__ void GATEGPU:: RYdagger(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE Block, STATE_T costheta, STATE_T sintheta)
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;           //thread number
    QSIZE BlockNum = idx / Block;
    QSIZE BlockInt = idx% Block;
    QSIZE realIdx = BlockNum * 2 * Block + BlockInt;
    QSIZE corIdx = realIdx + Block;
    if (corIdx < Dim)
    {
        STATE_T X1 = psireal[realIdx];
        STATE_T X2 = psireal[corIdx];
        STATE_T Y1 = psiimag[realIdx];
        STATE_T Y2 = psiimag[corIdx];
        psireal[realIdx] = X1*costheta + X2*sintheta;
        psireal[corIdx] = -X1*sintheta + X2*costheta;
        psiimag[realIdx] = Y1*costheta + Y2*sintheta;
        psiimag[corIdx] = Y2*costheta - Y1*sintheta;
    }
} //checked

__global__ void GATEGPU:: RZ(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE Block, STATE_T costheta, STATE_T sintheta)
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;

    QSIZE BlockNum = idx / Block;
    QSIZE BlockInt = idx% Block;
    QSIZE corIdx = BlockNum * 2 * Block + BlockInt + Block;
    if (corIdx < Dim)
    {
        STATE_T X2 = psireal[corIdx];
        STATE_T Y2 = psiimag[corIdx];
        psireal[corIdx] = X2*costheta - Y2*sintheta;
        psiimag[corIdx] = X2*sintheta + Y2*costheta;
    }
}

__global__ void GATEGPU:: RZdagger(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE Block, STATE_T costheta, STATE_T sintheta)
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;

    QSIZE BlockNum = idx / Block;
    QSIZE BlockInt = idx% Block;
    QSIZE corIdx = BlockNum * 2 * Block + BlockInt + Block;
    if (corIdx < Dim)
    {
        STATE_T X2 = psireal[corIdx];
        STATE_T Y2 = psiimag[corIdx];
        psireal[corIdx] = X2*costheta + Y2*sintheta;
        psiimag[corIdx] = -X2*sintheta + Y2*costheta;
    }
}

__global__ void GATEGPU:: NOT(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE Block)
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;           //thread number
    QSIZE BlockNum = idx / Block;
    QSIZE BlockInt = idx% Block;
    QSIZE realIdx = BlockNum * 2 * Block + BlockInt;
    QSIZE corIdx = realIdx + Block;
    if (corIdx < Dim)
    {
        STATE_T X1 = psireal[realIdx];
        STATE_T X2 = psireal[corIdx];
        STATE_T Y1 = psiimag[realIdx];
        STATE_T Y2 = psiimag[corIdx];
        psireal[realIdx] = X2;
        psireal[corIdx] = X1;
        psiimag[realIdx] = Y2;
        psiimag[corIdx] = Y1;
    }
}

__global__ void GATEGPU:: NOTdagger(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE Block)
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;           //thread number
    QSIZE BlockNum = idx / Block;
    QSIZE BlockInt = idx% Block;
    QSIZE realIdx = BlockNum * 2 * Block + BlockInt;
    QSIZE corIdx = realIdx + Block;
    if (corIdx < Dim)
    {
        STATE_T X1 = psireal[realIdx];
        STATE_T X2 = psireal[corIdx];
        STATE_T Y1 = psiimag[realIdx];
        STATE_T Y2 = psiimag[corIdx];
        psireal[realIdx] = X2;
        psireal[corIdx] = X1;
        psiimag[realIdx] = Y2;
        psiimag[corIdx] = Y1;
    }
}

__global__ void GATEGPU:: CNOT(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE Block1, QSIZE Block2)    //2^(qnum)           q9q8q7...q0 
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;
    QSIZE corIdx, realIdx;
    if (Block1 >  Block2)
    {
        corIdx = (idx / (Block1 / 2)) * 2 * Block1 + Block1 + (idx % (Block1 / 2) / Block2) * 2 * Block2 + idx%  Block2;
    }
    else
    {
        corIdx = (idx / (Block2 / 2)) * 2 * Block2 + (idx % (Block2 / 2) / Block1) * 2 * Block1 + Block1 + idx%  Block1;
    }
    realIdx = corIdx + Block2;
    if (realIdx < Dim)
    {
        STATE_T X1 = psireal[corIdx];                                   //10
        STATE_T Y1 = psiimag[corIdx];
        STATE_T X2 = psireal[realIdx];                                  //11
        STATE_T Y2 = psiimag[realIdx];
        psireal[corIdx] = X2;
        psiimag[corIdx] = Y2;
        psireal[realIdx] = X1;
        psiimag[realIdx] = Y1;
    }
}

__global__ void GATEGPU:: CNOTdagger(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE Block1, QSIZE Block2)    //2^(qnum)           q9q8q7...q0 
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;
    QSIZE corIdx, realIdx;
    if (Block1 >  Block2)
    {
        corIdx = (idx / (Block1 / 2)) * 2 * Block1 + Block1 + (idx % (Block1 / 2) / Block2) * 2 * Block2 + idx%  Block2;
    }
    else
    {
        corIdx = (idx / (Block2 / 2)) * 2 * Block2 + (idx % (Block2 / 2) / Block1) * 2 * Block1 + Block1 + idx%  Block1;
    }
    realIdx = corIdx + Block2;
    if (realIdx < Dim)
    {
        STATE_T X1 = psireal[corIdx];                                   //10
        STATE_T Y1 = psiimag[corIdx];
        STATE_T X2 = psireal[realIdx];                                  //11
        STATE_T Y2 = psiimag[realIdx];
        psireal[corIdx] = X2;
        psiimag[corIdx] = Y2;
        psireal[realIdx] = X1;
        psiimag[realIdx] = Y1;
    }
}

__global__ void GATEGPU:: CR(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE Block1, QSIZE Block2, STATE_T costheta, STATE_T sintheta)    //2^(qnum)           q9q8q7...q0 
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;
    QSIZE realIdx;
    if (Block1 >  Block2)
    {
        realIdx = (idx / (Block1 / 2)) * 2 * Block1 + Block1 + (idx % (Block1 / 2) / Block2) * 2 * Block2 + Block2 + idx%  Block2;
    }
    else
    {
        realIdx = (idx / (Block2 / 2)) * 2 * Block2 + Block2 + (idx % (Block2 / 2) / Block1) * 2 * Block1 + Block1 + idx%  Block1;
    }
    if (realIdx < Dim)
    {
        STATE_T X1 = psireal[realIdx];
        STATE_T Y1 = psiimag[realIdx];
        psireal[realIdx] = X1*costheta - Y1*sintheta;
        psiimag[realIdx] = X1*sintheta + Y1*costheta;
    }
}

__global__ void GATEGPU:: CRdagger(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE Block1, QSIZE Block2, STATE_T costheta, STATE_T sintheta)    //2^(qnum)           q9q8q7...q0 
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;
    QSIZE realIdx;
    if (Block1 >  Block2)
    {
        realIdx = (idx / (Block1 / 2)) * 2 * Block1 + Block1 + (idx % (Block1 / 2) / Block2) * 2 * Block2 + Block2 + idx%  Block2;
    }
    else
    {
        realIdx = (idx / (Block2 / 2)) * 2 * Block2 + Block2 + (idx % (Block2 / 2) / Block1) * 2 * Block1 + Block1 + idx%  Block1;
    }
    if (realIdx < Dim)
    {
        STATE_T X1 = psireal[realIdx];
        STATE_T Y1 = psiimag[realIdx];
        psireal[realIdx] = X1*costheta + Y1*sintheta;
        psiimag[realIdx] = -X1*sintheta + Y1*costheta;
    }
}

__global__ void GATEGPU:: iSWAP(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE Block1, QSIZE Block2)    //2^(qnum)           q9q8q7...q0 
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;
    QSIZE corIdx, realIdx, temp;
    if (Block1 <  Block2)
    {
        temp = Block1;
        Block1 = Block2;
        Block2 = temp;
    }
    corIdx = (idx / (Block1 / 2)) * 2 * Block1 + Block1 + (idx % (Block1 / 2) / Block2) * 2 * Block2 + idx%  Block2;
    realIdx = corIdx - Block1 + Block2;
    if (realIdx < Dim)
    {
        STATE_T X1 = psireal[corIdx];                                   //10
        STATE_T Y1 = psiimag[corIdx];
        STATE_T X2 = psireal[realIdx];                                  //11
        STATE_T Y2 = psiimag[realIdx];
        psireal[corIdx] = Y2;
        psiimag[corIdx] = -X2;
        psireal[realIdx] = Y1;
        psiimag[realIdx] = -X1;
    }
}

__global__ void GATEGPU:: iSWAPdagger(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE Block1, QSIZE Block2)    //2^(qnum)           q9q8q7...q0 
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;
    QSIZE corIdx, realIdx, temp;
    if (Block1 <  Block2)
    {
        temp = Block1;
        Block1 = Block2;
        Block2 = temp;
    }
    corIdx = (idx / (Block1 / 2)) * 2 * Block1 + Block1 + (idx % (Block1 / 2) / Block2) * 2 * Block2 + idx%  Block2;
    realIdx = corIdx - Block1 + Block2;
    if (realIdx < Dim)
    {
        STATE_T X1 = psireal[corIdx];                                   //10
        STATE_T Y1 = psiimag[corIdx];
        STATE_T X2 = psireal[realIdx];                                  //11
        STATE_T Y2 = psiimag[realIdx];
        psireal[corIdx] = -Y2;
        psiimag[corIdx] = X2;
        psireal[realIdx] = -Y1;
        psiimag[realIdx] = X1;
    }
}

__global__ void GATEGPU:: sqiSWAP(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE Block1, QSIZE Block2)    //2^(qnum)           q9q8q7...q0 
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;
    QSIZE corIdx, realIdx, temp;
    if (Block1 <  Block2)
    {
        temp = Block1;
        Block1 = Block2;
        Block2 = temp;
    }
    corIdx = (idx / (Block1 / 2)) * 2 * Block1 + Block1 + (idx % (Block1 / 2) / Block2) * 2 * Block2 + idx%  Block2;
    realIdx = corIdx - Block1 + Block2;
    if (realIdx < Dim)
    {
        STATE_T X1 = psireal[corIdx];                                   //10
        STATE_T Y1 = psiimag[corIdx];
        STATE_T X2 = psireal[realIdx];                                  //11
        STATE_T Y2 = psiimag[realIdx];
        psireal[corIdx] = (X1 + Y2)*SQ2;
        psiimag[corIdx] = (Y1 - X2)*SQ2;
        psireal[realIdx] = (X2 + Y1)*SQ2;
        psiimag[realIdx] = (Y2 - X1)*SQ2;
    }
}

__global__ void GATEGPU:: sqiSWAPdagger(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE Block1, QSIZE Block2)    //2^(qnum)           q9q8q7...q0 
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;
    QSIZE corIdx, realIdx, temp;
    if (Block1 <  Block2)
    {
        temp = Block1;
        Block1 = Block2;
        Block2 = temp;
    }
    corIdx = (idx / (Block1 / 2)) * 2 * Block1 + Block1 + (idx % (Block1 / 2) / Block2) * 2 * Block2 + idx%  Block2;
    realIdx = corIdx - Block1 + Block2;
    if (realIdx < Dim)
    {
        STATE_T X1 = psireal[corIdx];                                   //10
        STATE_T Y1 = psiimag[corIdx];
        STATE_T X2 = psireal[realIdx];                                  //11
        STATE_T Y2 = psiimag[realIdx];
        psireal[corIdx] = (X1 - Y2)*SQ2;
        psiimag[corIdx] = (Y1 + X2)*SQ2;
        psireal[realIdx] = (X2 - Y1)*SQ2;
        psiimag[realIdx] = (Y2 + X1)*SQ2;
    }
}

__global__ void GATEGPU:: qubitprob(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE Block, STATE_T *pr)
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;           //thread number
    QSIZE bid = blockIdx.x, tid = threadIdx.x;
    QSIZE BlockNum = idx / Block;
    QSIZE BlockInt = idx% Block;
    QSIZE realIdx = BlockNum * 2 * Block + BlockInt;
    QSIZE corIdx = realIdx + Block;
    extern __shared__ STATE_T  dprob[];
    dprob[tid] = 0;
    int i;
    /*
    for (i = bid * blockDim.x + tid; i < Dim / 2; i += gridDim.x * THREADDIM)
    {
        //        QSIZE idx = bid*(psigpu.qnum-1))/THREADDIM.x + tid;
        //        QSIZE corIdx = idx / Block * 2 * Block + idx%Block;
        dprob[tid] += psireal[realIdx] * psireal[realIdx] + psiimag[realIdx] * psiimag[realIdx];
    }
    */
    //dprob[tid] = psireal[realIdx] * psireal[realIdx] + psiimag[realIdx] * psiimag[realIdx];

    //ͬ�� ��֤ÿ�� thread ���Ѿ��ѽ��д�� shared[tid] ����
    if (corIdx < Dim)
    {
        dprob[tid] = psireal[realIdx] * psireal[realIdx] + psiimag[realIdx] * psiimag[realIdx];   //��ʡ��?
        __syncthreads();//��״�ӷ�
        int offset = 1, mask = 1;
        while (offset < THREADDIM)
        {
            if ((tid & mask) == 0)
            {
                dprob[tid] += dprob[tid + offset];
            }
            offset += offset;
            mask = offset + mask;
            __syncthreads();
        }
        //����ʱ��,��¼���,ֻ�� thread 0���� threadIdx.x =
        //dprob[0]��ͼ��õ����ĸ���
        if (tid == 0)
        {
            pr[bid] = dprob[0];
        }
    }
}//checked
__global__ void GATEGPU::probsumnew1(STATE_T * psireal, STATE_T * psiimag, STATE_T *probtemp, size_t num1, size_t m, size_t Dim, size_t * block)
{
    size_t idx = blockDim.x*blockIdx.x + threadIdx.x;           //thread number
    size_t bid = blockIdx.x, tid = threadIdx.x;
    size_t index1, index = 0, index2, k, s;
    double temp = 0;
    index1 = num1 + idx;                              //index1��ʾidx��Ӧ�Ĳ���Ȩ��̬���
    if (index1 < (1u << m))
    {
        for (size_t j = 0; j < m; j++)
        {
            index += block[j] * ((index1 >> j) % 2);
        }//index ��ʾidx��Ӧ��̬�����
        for (size_t i = 0; i < Dim / (1u << m); i++)
        {
            index2 = i;
            for (size_t j = 0; j < m; j++)
            {
                s = index2 / block[j];
                k = index2 % block[j];
                index2 = s * 2 * block[j] + k;
            }
            index2 += index;
            temp += psireal[index2] * psireal[index2] + psiimag[index2] * psiimag[index2];
        }
        probtemp[idx] = temp;
    }
}
__global__ void GATEGPU:: probsum(STATE_T * pr, STATE_T * prob)
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;           //thread number
    if (0 == idx)
    {
        STATE_T dprob = 0;
        for (int i = 0; i < gridDim.x; i++)
        {
            dprob += pr[i];
        }
        *prob = dprob;
    }
}//checked and can be optimized

__global__ void GATEGPU:: qubitcollapse0(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE Block, STATE_T coef)
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;           //thread number
    QSIZE BlockNum = idx / Block;
    QSIZE BlockInt = idx% Block;
    QSIZE realIdx = BlockNum * 2 * Block + BlockInt;
    QSIZE corIdx = realIdx + Block;
    if (corIdx < Dim)
    {
        STATE_T X1 = psireal[realIdx];
        STATE_T Y1 = psiimag[realIdx];
        psireal[realIdx] = X1*coef;
        psireal[corIdx] = 0;
        psiimag[realIdx] = Y1*coef;
        psiimag[corIdx] = 0;
    }
}//checked

__global__ void GATEGPU:: qubitcollapse1(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE Block, STATE_T coef)
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;           //thread number
    QSIZE BlockNum = idx / Block;
    QSIZE BlockInt = idx% Block;
    QSIZE realIdx = BlockNum * 2 * Block + BlockInt;
    QSIZE corIdx = realIdx + Block;
    if (corIdx < Dim)
    {
        STATE_T X2 = psireal[corIdx];
        STATE_T Y2 = psiimag[corIdx];
        psireal[realIdx] = 0;
        psireal[corIdx] = X2*coef;
        psiimag[realIdx] = 0;
        psiimag[corIdx] = Y2*coef;
    }
}//checked

__global__ void GATEGPU:: controlHadamard(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE * block1, QSIZE  block2, QSIZE m)
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;           //thread number

                                                                      //QSIZE BlockNum = idx / Block;
                                                                      //QSIZE BlockInt = idx% Block;    
    QSIZE i, j, k;
    if (idx < Dim / (1 << m))
    {
        QSIZE index = idx;
        for (i = 0; i < m; i++)
        {
            j = index / block1[i];
            k = index % block1[i];
            index = j * 2 * block1[i] + block1[i] + k;

        }
        QSIZE corIdx = index;                                   //1111
        QSIZE realIdx = corIdx - block2;                    //1110
        STATE_T X1 = psireal[realIdx];
        STATE_T X2 = psireal[corIdx];
        STATE_T Y1 = psiimag[realIdx];
        STATE_T Y2 = psiimag[corIdx];
        psireal[realIdx] = (X1 + X2)*SQ2;
        psireal[corIdx] = (X1 - X2)*SQ2;
        psiimag[realIdx] = (Y1 + Y2)*SQ2;
        psiimag[corIdx] = (Y1 - Y2)*SQ2;
    }
}//checked

__global__ void GATEGPU:: controlHadamarddagger(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE * block1, QSIZE  block2, QSIZE m)
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;           //thread number

                                                                      //QSIZE BlockNum = idx / Block;
                                                                      //QSIZE BlockInt = idx% Block;    
    QSIZE i, j, k;
    if (idx < Dim / (1 << m))
    {
        QSIZE index = idx;
        for (i = 0; i < m; i++)
        {
            j = index / block1[i];
            k = index % block1[i];
            index = j * 2 * block1[i] + block1[i] + k;

        }
        QSIZE corIdx = index;                                   //1111
        QSIZE realIdx = corIdx - block2;                    //1110
        STATE_T X1 = psireal[realIdx];
        STATE_T X2 = psireal[corIdx];
        STATE_T Y1 = psiimag[realIdx];
        STATE_T Y2 = psiimag[corIdx];
        psireal[realIdx] = (X1 + X2)*SQ2;
        psireal[corIdx] = (X1 - X2)*SQ2;
        psiimag[realIdx] = (Y1 + Y2)*SQ2;
        psiimag[corIdx] = (Y1 - Y2)*SQ2;
    }
}//checked

__global__ void GATEGPU:: controlRX(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE * block1, QSIZE  block2, QSIZE m, STATE_T costheta, STATE_T sintheta)
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;           //thread number

                                                                      //QSIZE BlockNum = idx / Block;
                                                                      //QSIZE BlockInt = idx% Block;    
    QSIZE i, j, k;
    if (idx < Dim / (1 << m))
    {
        QSIZE index = idx;
        for (i = 0; i < m; i++)
        {
            j = index / block1[i];
            k = index % block1[i];
            index = j * 2 * block1[i] + block1[i] + k;

        }
        QSIZE corIdx = index;                                   //1111
        QSIZE realIdx = corIdx - block2;                    //1110
        STATE_T X1 = psireal[realIdx];
        STATE_T X2 = psireal[corIdx];
        STATE_T Y1 = psiimag[realIdx];
        STATE_T Y2 = psiimag[corIdx];
        psireal[realIdx] = X1*costheta + Y2*sintheta;
        psireal[corIdx] = Y1*sintheta + X2*costheta;
        psiimag[realIdx] = Y1*costheta - X2*sintheta;
        psiimag[corIdx] = Y2*costheta - X1*sintheta;
    }
}       //checked

__global__ void GATEGPU:: controlRXdagger(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE * block1, QSIZE  block2, QSIZE m, STATE_T costheta, STATE_T sintheta)
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;           //thread number

                                                                      //QSIZE BlockNum = idx / Block;
                                                                      //QSIZE BlockInt = idx% Block;    
    QSIZE i, j, k;
    if (idx < Dim / (1 << m))
    {
        QSIZE index = idx;
        for (i = 0; i < m; i++)
        {
            j = index / block1[i];
            k = index % block1[i];
            index = j * 2 * block1[i] + block1[i] + k;

        }
        QSIZE corIdx = index;                                   //1111
        QSIZE realIdx = corIdx - block2;                    //1110
        STATE_T X1 = psireal[realIdx];
        STATE_T X2 = psireal[corIdx];
        STATE_T Y1 = psiimag[realIdx];
        STATE_T Y2 = psiimag[corIdx];
        psireal[realIdx] = X1*costheta - Y2*sintheta;
        psireal[corIdx] = -Y1*sintheta + X2*costheta;
        psiimag[realIdx] = Y1*costheta + X2*sintheta;
        psiimag[corIdx] = Y2*costheta + X1*sintheta;
    }
}       //checked

__global__ void GATEGPU:: controlRY(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE * block1, QSIZE  block2, QSIZE m, STATE_T costheta, STATE_T sintheta)
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;           //thread number

                                                                      //QSIZE BlockNum = idx / Block;
                                                                      //QSIZE BlockInt = idx% Block;    
    QSIZE i, j, k;
    if (idx < Dim / (1 << m))
    {
        QSIZE index = idx;
        for (i = 0; i < m; i++)
        {
            j = index / block1[i];
            k = index % block1[i];
            index = j * 2 * block1[i] + block1[i] + k;

        }
        QSIZE corIdx = index;                                   //1111
        QSIZE realIdx = corIdx - block2;                    //1110
        STATE_T X1 = psireal[realIdx];
        STATE_T X2 = psireal[corIdx];
        STATE_T Y1 = psiimag[realIdx];
        STATE_T Y2 = psiimag[corIdx];
        psireal[realIdx] = X1*costheta - X2*sintheta;
        psireal[corIdx] = X1*sintheta + X2*costheta;
        psiimag[realIdx] = Y1*costheta - Y2*sintheta;
        psiimag[corIdx] = Y2*costheta + Y1*sintheta;
    }
}       //checked

__global__ void GATEGPU:: controlRYdagger(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE * block1, QSIZE  block2, QSIZE m, STATE_T costheta, STATE_T sintheta)
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;           //thread number

                                                                      //QSIZE BlockNum = idx / Block;
                                                                      //QSIZE BlockInt = idx% Block;    
    QSIZE i, j, k;
    if (idx < Dim / (1 << m))
    {
        QSIZE index = idx;
        for (i = 0; i < m; i++)
        {
            j = index / block1[i];
            k = index % block1[i];
            index = j * 2 * block1[i] + block1[i] + k;

        }
        QSIZE corIdx = index;                                   //1111
        QSIZE realIdx = corIdx - block2;                    //1110
        STATE_T X1 = psireal[realIdx];
        STATE_T X2 = psireal[corIdx];
        STATE_T Y1 = psiimag[realIdx];
        STATE_T Y2 = psiimag[corIdx];
        psireal[realIdx] = X1*costheta + X2*sintheta;
        psireal[corIdx] = -X1*sintheta + X2*costheta;
        psiimag[realIdx] = Y1*costheta + Y2*sintheta;
        psiimag[corIdx] = Y2*costheta - Y1*sintheta;
    }
}       //checked

__global__ void GATEGPU:: controlRZ(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE * block1, QSIZE  block2, QSIZE m, STATE_T costheta, STATE_T sintheta)
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;           //thread number
    QSIZE i, j, k;
    if (idx < Dim / (1 << m))
    {
        QSIZE index = idx;
        for (i = 0; i < m; i++)
        {
            j = index / block1[i];
            k = index % block1[i];
            index = j * 2 * block1[i] + block1[i] + k;

        }
        QSIZE corIdx = index;                                   //1111        
        STATE_T X2 = psireal[corIdx];
        STATE_T Y2 = psiimag[corIdx];
        psireal[corIdx] = X2*costheta - Y2*sintheta;
        psiimag[corIdx] = X2*sintheta + Y2*costheta;
    }
}       //checked

__global__ void GATEGPU:: controlRZdagger(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE * block1, QSIZE  block2, QSIZE m, STATE_T costheta, STATE_T sintheta)
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;           //thread number
    QSIZE i, j, k;
    if (idx < Dim / (1 << m))
    {
        QSIZE index = idx;
        for (i = 0; i < m; i++)
        {
            j = index / block1[i];
            k = index % block1[i];
            index = j * 2 * block1[i] + block1[i] + k;

        }
        QSIZE corIdx = index;                                   //1111        
        STATE_T X2 = psireal[corIdx];
        STATE_T Y2 = psiimag[corIdx];
        psireal[corIdx] = X2*costheta + Y2*sintheta;
        psiimag[corIdx] = -X2*sintheta + Y2*costheta;
    }
}       //checked

__global__ void GATEGPU:: toffoli(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE  block0, QSIZE  block1, QSIZE block2)
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;           //thread number
    QSIZE block = block2, temp;
    if (block0 > block1)
    {
        temp = block1;
        block1 = block0;
        block0 = temp;
    }
    if (block0 > block2)
    {
        temp = block2;
        block2 = block0;
        block0 = temp;
    }
    if (block1 > block2)
    {
        temp = block2;
        block2 = block1;
        block1 = temp;
    }                           //small to large sequence
    if (idx < Dim / (1 << 3))
    {
        QSIZE corIdx = idx / block0 * 2 * block0 + block0 + idx%block0, realIdx;
        corIdx = corIdx / block1 * 2 * block1 + block1 + corIdx%block1;
        corIdx = corIdx / block2 * 2 * block2 + block2 + corIdx%block2;
        realIdx = corIdx - block;
        STATE_T X1 = psireal[realIdx];
        STATE_T X2 = psireal[corIdx];
        STATE_T Y1 = psiimag[realIdx];
        STATE_T Y2 = psiimag[corIdx];
        psireal[realIdx] = X2;
        psireal[corIdx] = X1;
        psiimag[realIdx] = Y2;
        psiimag[corIdx] = Y1;
    }
}     //checked

__global__ void GATEGPU:: toffolidagger(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE  block0, QSIZE  block1, QSIZE block2)
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;           //thread number
    QSIZE block = block2, temp;
    if (block0 > block1)
    {
        temp = block1;
        block1 = block0;
        block0 = temp;
    }
    if (block0 > block2)
    {
        temp = block2;
        block2 = block0;
        block0 = temp;
    }
    if (block1 > block2)
    {
        temp = block2;
        block2 = block1;
        block1 = temp;
    }                           //small to large sequence
    if (idx < Dim / (1 << 3))
    {
        QSIZE corIdx = idx / block0 * 2 * block0 + block0 + idx%block0, realIdx;
        corIdx = corIdx / block1 * 2 * block1 + block1 + corIdx%block1;
        corIdx = corIdx / block2 * 2 * block2 + block2 + corIdx%block2;
        realIdx = corIdx - block;
        STATE_T X1 = psireal[realIdx];
        STATE_T X2 = psireal[corIdx];
        STATE_T Y1 = psiimag[realIdx];
        STATE_T Y2 = psiimag[corIdx];
        psireal[realIdx] = X2;
        psireal[corIdx] = X1;
        psiimag[realIdx] = Y2;
        psiimag[corIdx] = Y1;
    }
}     //checked
/*
__global__ void GATEGPU:: qbReset(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, QSIZE Block)      //reset to |0>, this operator is NOT UNITARY
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;           //thread number
    QSIZE BlockNum = idx / Block;
    QSIZE BlockInt = idx% Block;
    QSIZE realIdx = BlockNum * 2 * Block + BlockInt;
    QSIZE corIdx = realIdx + Block;

    if (corIdx < Dim)
    {
        STATE_T X1 = psireal[realIdx];
        STATE_T Y1 = psiimag[realIdx];
        psireal[realIdx] = X1;
        psireal[corIdx] = 0;
        psiimag[realIdx] = Y1;
        psiimag[corIdx] = 0;
    }
}
*/


/**************************************************************************************
psireal:
psiimag:
pro:      save probability
block:    qubit number
m:        target qubit number
dec:      target qubit state





****************************************************************************************/
__global__ void GATEGPU:: multiprob(STATE_T * psireal, STATE_T * psiimag,QSIZE Dim, STATE_T * pro, QSIZE * block, QSIZE m, QSIZE dec)
{
    QSIZE idx = blockDim.x*blockIdx.x + threadIdx.x;           //thread number
    QSIZE bid = blockIdx.x, tid = threadIdx.x;
    //QSIZE BlockNum = idx / Block;
    //QSIZE BlockInt = idx% Block;    
    extern __shared__ STATE_T dprob[];
    dprob[tid] = 0;
    QSIZE i, j, k;
    if (idx < Dim / (1 << m))
    {
        QSIZE index = idx;
        for (i = 0; i < m; i++)
        {
            j = index / block[i];
            k = index % block[i];
            index = j * 2 * block[i] + k;
        }                                                              //index Ŀ�����ȫΪ0
        QSIZE realIdx = index + dec;                                   //��Ҫ�ӵ�̬�ĸ���
        dprob[tid] = psireal[realIdx] * psireal[realIdx] + psiimag[realIdx] * psiimag[realIdx];
        __syncthreads();//��״�ӷ�
        int offset = 1, mask = 1;
        while (offset < THREADDIM)
        {
            if ((tid & mask) == 0)
            {
                dprob[tid] += dprob[tid + offset];
            }
            offset += offset;
            mask = offset + mask;
            __syncthreads();
        }
        //����ʱ��,��¼���,ֻ�� thread 0���� threadIdx.x =
        //dprob[0]��ͼ��õ����ĸ���
        if (tid == 0)
        {
            pro[bid] = dprob[0];                       //�ټ���pro�ĺ;͵õ����ĸ���
        }
    }
}


/***********************************************************************
Copyright:
Author:Xue Cheng
Date:2017-12-13
Description: Definition of Encapsulation of GPU gates
************************************************************************/

#define SET_BLOCKDIM  BLOCKDIM = (1 << (psigpu.qnum - 1)) / THREADDIM;

int GATEGPU:: devicecount()
{
    int count;
    hipGetDeviceCount(&count);
    return count;
}


bool getSynchronizeResult(hipError_t cudaStatue, char * pcGate)
{

    if (hipSuccess != cudaStatue)
    {
        cout << "err " << pcGate << " = " << hipGetErrorString(cudaStatue) << endl;
        return false;
    }
    return true;
}

#define GET_SYN_RES(x)      hipError_t cudaStatue = hipDeviceSynchronize();\
                            return getSynchronizeResult(cudaStatue,(x));

bool GATEGPU::destroyState(QState& psi, QState& psigpu,size_t stQnum)
{

    if ((nullptr == psi.real) ||
        (nullptr == psi.imag) ||
        (nullptr == psigpu.real) ||
        (nullptr == psigpu.imag))
    {
        return false;
    }

    if (stQnum<30)
    {
        hipError_t cudaStates = hipFree(psigpu.real);
        if (hipSuccess != cudaStates)
        {
            cout << "psigpu.real free error" << endl;
            return false;
        }
        cudaStates =hipFree(psigpu.imag);
        if (hipSuccess != cudaStates)
        {
            cout << "psigpu.imag free error" << endl;
            return false;
        }
        free(psi.real);
        free(psi.imag);
    }
    else
    {
        hipHostFree(psigpu.real);
        hipHostFree(psigpu.imag);
    }


    return true;
}

bool GATEGPU::clearState(QState& psi, QState& psigpu,size_t stQnum)
{

    if ((nullptr == psi.real) ||
        (nullptr == psi.imag) ||
        (nullptr == psigpu.real) ||
        (nullptr == psigpu.imag))
    {
        return false;
    }

    if (stQnum<30)
    {
        QSIZE qsDim = (1ll << stQnum);
        memset(psi.real,0, qsDim *sizeof(STATE_T));
        memset(psi.imag, 0, qsDim * sizeof(STATE_T));
        psi.real[0] = 1;

        //hipFree(psigpu.real);
        //hipFree(psigpu.imag);
        hipError_t cudaStatue = hipMemcpy(psigpu.real, psi.real, sizeof(STATE_T)*qsDim, hipMemcpyHostToDevice);
        if (hipSuccess != cudaStatue)
        {
            cout << "psigpu real memcpy error" << endl;
        }
        cudaStatue = hipMemcpy(psigpu.imag, psi.imag, sizeof(STATE_T)*qsDim, hipMemcpyHostToDevice);
        if (hipSuccess != cudaStatue)
        {
            cout << "psigpu imag memcpy error" << endl;
        }
    }
    else
    {
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        initState << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag, 1 << (psigpu.qnum));
    }

    return true;
}

bool GATEGPU::initstate(QState& psi, QState& psigpu, int qnum)
{
    //QState psigpu;
    if (qnum >= 30)
    {
        hipError_t cudaStatus = hipHostAlloc(&psi.real, sizeof(double)*(1ll << qnum), hipHostMallocMapped);
        if (cudaStatus != hipSuccess)
        {
            printf("host alloc fail!\n");
            return false;
        }
        hipError_t cudaStatus1 = hipHostAlloc(&psi.imag, sizeof(double)*(1ll << qnum), hipHostMallocMapped);
        if (cudaStatus1 != hipSuccess)
        {
            printf("host alloc fail!\n");
            return false;
        }
        hipHostGetDevicePointer(&psigpu.real, psi.real, 0);
        hipHostGetDevicePointer(&psigpu.imag, psi.imag, 0);
        psi.qnum = qnum;
        psigpu.qnum = qnum;
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
            initState << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag, 1 << (psigpu.qnum));
        //std::cout << "err = " << hipGetErrorString(hipDeviceSynchronize()) << endl;
    }
    else 
    {
        QSIZE Dim = 1 << qnum;
        hipError_t cudaStatus;
        psi.real = (STATE_T*)malloc(Dim * sizeof(STATE_T));
        if (nullptr == psi.real)
        {
            printf("psi.real alloc memory error\n");
            return false;
        }
        psi.imag = (STATE_T*)malloc(Dim * sizeof(STATE_T));
        if (nullptr == psi.real)
        {
            printf("psi.imag alloc memory error\n");
            free(psi.imag);
            return false;
        }
        cudaStatus = hipMalloc((void**)&psigpu.real, sizeof(STATE_T)*Dim);
        if (hipSuccess != cudaStatus)
        {
            printf("psigpu.real alloc gpu memoery error!\n");
            free(psi.real);
            free(psi.imag);
            return false;
        }
        cudaStatus = hipMalloc((void**)&psigpu.imag, sizeof(STATE_T)*Dim);
        if (hipSuccess != cudaStatus)
        {
            printf("psigpu.imag alloc gpu memoery error!\n");
            free(psi.real);
            free(psi.imag);
            hipFree(psigpu.real);
            return false;
        }
        
        memset(psi.real,0,Dim * sizeof(STATE_T));
        memset(psi.imag, 0, Dim * sizeof(STATE_T));
        psi.real[0] = 1;
        
        cudaStatus = hipMemcpy(psigpu.real, psi.real, sizeof(STATE_T)*Dim, hipMemcpyHostToDevice);
        if (hipSuccess != cudaStatus)
        {
            printf("psigpu.imag alloc gpu memoery error!\n");
            free(psi.real);
            free(psi.imag);
            hipFree(psigpu.real);
            hipFree(psigpu.imag);
            return false;
        }

        cudaStatus = hipMemcpy(psigpu.imag, psi.imag, sizeof(STATE_T)*Dim, hipMemcpyHostToDevice);
        if (hipSuccess != cudaStatus)
        {
            printf("psigpu.imag alloc gpu memoery error!\n");
            free(psi.real);
            free(psi.imag);
            hipFree(psigpu.real);
            hipFree(psigpu.imag);
            return false;
        }
        psigpu.qnum = qnum;
        psi.qnum = qnum;
    }

    return true;
}



bool GATEGPU:: Hadamard(QState& psigpu, size_t qn, double error_rate)

{
    if (randGenerator() > error_rate)
    {
        //QState* QPsi = (QState*)psi;
        //int BLOCKDIM = (1 << (psigpu.qnum - 1)) / THREADDIM;
        QSIZE BLOCKDIM = (1 << (psigpu.qnum - 1)) / THREADDIM;
        Hadamard << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag, 1 << (psigpu.qnum), 1 << qn);
    }
    //GET_SYN_RES("Hadamard");
    hipError_t cudaStatue = hipDeviceSynchronize(); 
    return getSynchronizeResult(cudaStatue, "Hadamard");
}
bool GATEGPU::Hadamardnew(QState& psigpu, size_t qn, double error_rate)

{
    if (randGenerator() > error_rate)
    {
        //QState* QPsi = (QState*)psi;
        //int BLOCKDIM = (1 << (psigpu.qnum - 1)) / THREADDIM;
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        Hadamardnew << < 4096, THREADDIM >> > (psigpu.real, psigpu.imag, 1 << (psigpu.qnum), 1 << qn);
    }
    //std::cout << "err = " << hipGetErrorString(hipDeviceSynchronize()) << endl;
    GET_SYN_RES("Hadamardnew")
}
bool GATEGPU:: Hadamarddagger(QState& psigpu, size_t qn, double error_rate)
{
    if (randGenerator() > error_rate)
    {
        //QState* QPsi = (QState*)psi;
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        Hadamarddagger << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), 1 << qn);
        std::cout << "err Hadamarddagger= " << hipGetErrorString(hipDeviceSynchronize()) << endl;
    }
   // std::cout << "err = " << hipGetErrorString(hipDeviceSynchronize()) << endl;
    GET_SYN_RES("Hadamarddagger")

}

bool GATEGPU:: RX(QState& psigpu, size_t qn, double theta, double error_rate)
{
    if (randGenerator() > error_rate)
    {
        //QState* QPsi = (QState*)psi;
        double costheta = cos(theta / 2), sintheta = sin(theta / 2);
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        RX << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), 1 << qn, costheta, sintheta);
    }

    GET_SYN_RES("RX")
}
bool GATEGPU:: RXdagger(QState& psigpu, size_t qn, double theta, double error_rate)
{
    if (randGenerator() > error_rate)
    {
        //QState* QPsi = (QState*)psi;
        double costheta = cos(theta / 2), sintheta = sin(theta / 2);
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        RXdagger << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), 1 << qn, costheta, sintheta);
        //hipDeviceSynchronize();
    }
    GET_SYN_RES("RXdagger")
}
bool GATEGPU::RY(QState& psigpu, size_t qn, double theta, double error_rate)
{
    if (randGenerator() > error_rate)
    {
        //QState* QPsi = (QState*)psi;
        double costheta = cos(theta / 2), sintheta = sin(theta / 2);
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        RY << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag, 1 << (psigpu.qnum), 1 << qn, costheta, sintheta);
        //hipDeviceSynchronize();
        //std::cout << "errry = " << hipGetErrorString(hipDeviceSynchronize()) << endl;
    }
    GET_SYN_RES("RY")
}
bool GATEGPU:: RYdagger(QState& psigpu, size_t qn, double theta, double error_rate)
{
    if (randGenerator() > error_rate)
    {
        //QState* QPsi = (QState*)psi;
        double costheta = cos(theta / 2), sintheta = sin(theta / 2);
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        RYdagger << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), 1 << qn, costheta, sintheta);
        //hipDeviceSynchronize();
    }
    GET_SYN_RES("RYdagger")
}
bool GATEGPU:: RZ(QState& psigpu, size_t qn, double theta, double error_rate)
{
    if (randGenerator() > error_rate)
    {
        //QState* QPsi = (QState*)psi;
        double costheta = cos(theta), sintheta = sin(theta);
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        RZ << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), 1 << qn, costheta, sintheta);
        //std::cout << "err rz= " << hipGetErrorString(hipDeviceSynchronize()) << endl;
        //hipDeviceSynchronize();
    }
    GET_SYN_RES("RZ")
}

bool GATEGPU:: RZdagger(QState& psigpu, size_t qn, double theta, double error_rate)
{
    if (randGenerator() > error_rate)
    {
        //QState* QPsi = (QState*)psi;
        double costheta = cos(theta), sintheta = sin(theta);
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        RZdagger << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), 1 << qn, costheta, sintheta);
        //hipDeviceSynchronize();
    }
    GET_SYN_RES("RZdagger")
}
bool GATEGPU:: NOT(QState& psigpu, size_t qn, double error_rate)
{
    if (randGenerator() > error_rate)
    {
        //QState* QPsi = (QState*)psi;
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        NOT << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), 1 << qn);
        //std::cout << "err NOT= " << hipGetErrorString(hipDeviceSynchronize()) << endl;
    }
    GET_SYN_RES("NOT")
}
bool GATEGPU:: NOTdagger(QState& psigpu, size_t qn, double error_rate)
{
    if (randGenerator() > error_rate)
    {
        //QState* QPsi = (QState*)psi;
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        NOTdagger << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), 1 << qn);
        //hipDeviceSynchronize();
    }
    GET_SYN_RES("NOTdagger")
}

bool GATEGPU:: CNOT(QState& psigpu, size_t qn0, size_t qn1, double error_rate)
{
    if (randGenerator() > error_rate)
    {
        //QState* QPsi = (QState*)psi;
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        CNOT << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), 1 << qn0, 1 << qn1);
        //std::cout << "err CNOT= " << hipGetErrorString(hipDeviceSynchronize()) << endl;
    }
    GET_SYN_RES("CNOT")
}
bool GATEGPU::CNOTdagger(QState& psigpu, size_t qn0, size_t qn1, double error_rate)
{
    if (randGenerator() > error_rate)
    {
        //QState* QPsi = (QState*)psi;
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        CNOTdagger << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag, 1 << (psigpu.qnum), 1 << qn0, 1 << qn1);
    }
    GET_SYN_RES("CNOTdagger")
}
bool GATEGPU:: CR(QState& psigpu, size_t qn0, size_t qn1, double theta, double error_rate)
{
    if (randGenerator() > error_rate)
    {
        //QState* QPsi = (QState*)psi;
        double costheta = cos(theta), sintheta = sin(theta);
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        CR << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), 1 << qn0, 1 << qn1, costheta, sintheta);
        //std::cout << "err CR= " << hipGetErrorString(hipDeviceSynchronize()) << endl;
    }
    GET_SYN_RES("CR")
}
bool GATEGPU:: CRdagger(QState& psigpu, size_t qn0, size_t qn1, double theta, double error_rate)
{
    if (randGenerator() > error_rate)
    {
        //QState* QPsi = (QState*)psi;
        double costheta = cos(theta), sintheta = sin(theta);
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        CRdagger << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), 1 << qn0, 1 << qn1, costheta, sintheta);
    }
    GET_SYN_RES("CRdagger")
}
bool GATEGPU:: iSWAP(QState& psigpu, size_t qn0, size_t qn1, double error_rate)
{
    if (randGenerator() > error_rate)
    {
        //QState* QPsi = (QState*)psi;
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        iSWAP << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), 1 << qn0, 1 << qn1);
        //std::cout << "err iSWAP= " << hipGetErrorString(hipDeviceSynchronize()) << endl;
    }
    GET_SYN_RES("iSWAP")
}
bool GATEGPU:: iSWAPdagger(QState& psigpu, size_t qn0, size_t qn1, double error_rate)
{
    if (randGenerator() > error_rate)
    {
        //QState* QPsi = (QState*)psi;
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        iSWAPdagger << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), 1 << qn0, 1 << qn1);
    }
    GET_SYN_RES("iSWAPdagger")
}
bool GATEGPU:: sqiSWAP(QState& psigpu, size_t qn0, size_t qn1, double error_rate)
{
    if (randGenerator() > error_rate)
    {
        //QState* QPsi = (QState*)psi;
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        sqiSWAP << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), 1 << qn0, 1 << qn1);
        //std::cout << "err sqiSWAP= " << hipGetErrorString(hipDeviceSynchronize()) << endl;
    }
    GET_SYN_RES("sqiSWAP")
}
bool GATEGPU:: sqiSWAPdagger(QState& psigpu, size_t qn0, size_t qn1, double error_rate)
{
    if (randGenerator() > error_rate)
    {
        //QState* QPsi = (QState*)psi;
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        sqiSWAPdagger << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), 1 << qn0, 1 << qn1);
    }
    GET_SYN_RES("sqiSWAP")
}
bool GATEGPU:: controlHadamard(QState& psigpu, Qnum& qnum, double error_rate)
{
    if (randGenerator() > error_rate)
    {
        QSIZE m = qnum.size();
        QSIZE target = qnum.back();
        sort(qnum.begin(), qnum.end());
        QSIZE *block, *blockgpu;
        hipHostAlloc((void **)&block, sizeof(QSIZE)*m, hipHostMallocMapped);
        hipHostGetDevicePointer(&blockgpu, block, 0);
        for (QSIZE i = 0; i < m; i++)
        {
            block[i] = 1 << qnum[i];
        }
        //QState* QPsi = (QState*)psi;
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        controlHadamard << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), blockgpu, 1 << target, m);
        hipHostFree(block);
        //std::cout << "err = " << hipGetErrorString(hipDeviceSynchronize()) << endl;
    }
    GET_SYN_RES("controlHadamard")
}
bool GATEGPU:: controlHadamarddagger(QState& psigpu, Qnum& qnum, double error_rate)
{
    if (randGenerator() > error_rate)
    {
        QSIZE m = qnum.size();
        QSIZE target = qnum.back();
        sort(qnum.begin(), qnum.end());
        QSIZE *block, *blockgpu;
        hipHostAlloc((void **)&block, sizeof(QSIZE)*m, hipHostMallocMapped);
        hipHostGetDevicePointer(&blockgpu, block, 0);
        for (QSIZE i = 0; i < m; i++)
        {
            block[i] = 1 << qnum[i];
        }
        //QState* QPsi = (QState*)psi;
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        controlHadamarddagger << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), blockgpu, 1 << target, m);
        hipHostFree(block);
    }
    GET_SYN_RES("controlHadamarddagger")
}
bool GATEGPU:: controlRX(QState& psigpu, Qnum& qnum, double theta, double error_rate)
{
    if (randGenerator() > error_rate)
    {
        double costheta = cos(theta / 2), sintheta = sin(theta / 2);
        QSIZE m = qnum.size();
        QSIZE target = qnum.back();
        sort(qnum.begin(), qnum.end());
        QSIZE *block, *blockgpu;
        hipHostAlloc((void **)&block, sizeof(QSIZE)*m, hipHostMallocMapped);
        hipHostGetDevicePointer(&blockgpu, block, 0);
        for (QSIZE i = 0; i < m; i++)
        {
            block[i] = 1 << qnum[i];
        }
        //QState* QPsi = (QState*)psi;
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        controlRX << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), blockgpu, 1 << target, m, costheta, sintheta);
        hipHostFree(block);

    }
    GET_SYN_RES("controlRX")
}
bool GATEGPU:: controlRXdagger(QState& psigpu, Qnum& qnum, double theta, double error_rate)
{
    if (randGenerator() > error_rate)
    {
        double costheta = cos(theta / 2), sintheta = sin(theta / 2);
        QSIZE m = qnum.size();
        QSIZE target = qnum.back();
        sort(qnum.begin(), qnum.end());
        QSIZE *block, *blockgpu;
        hipHostAlloc((void **)&block, sizeof(QSIZE)*m, hipHostMallocMapped);
        hipHostGetDevicePointer(&blockgpu, block, 0);
        for (QSIZE i = 0; i < m; i++)
        {
            block[i] = 1 << qnum[i];
        }
        //QState* QPsi = (QState*)psi;
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        controlRXdagger << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), blockgpu, 1 << target, m, costheta, sintheta);
        hipHostFree(block);
    }
    GET_SYN_RES("controlRX")
}
bool GATEGPU:: controlRY(QState& psigpu, Qnum& qnum, double theta, double error_rate)
{
    if (randGenerator() > error_rate)
    {
        double costheta = cos(theta / 2), sintheta = sin(theta / 2);
        QSIZE m = qnum.size();
        QSIZE target = qnum.back();
        sort(qnum.begin(), qnum.end());
        QSIZE *block, *blockgpu;
        hipHostAlloc((void **)&block, sizeof(QSIZE)*m, hipHostMallocMapped);
        hipHostGetDevicePointer(&blockgpu, block, 0);
        for (QSIZE i = 0; i < m; i++)
        {
            block[i] = 1 << qnum[i];
        }
        //QState* QPsi = (QState*)psi;
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        controlRY << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), blockgpu, 1 << target, m, costheta, sintheta);
        hipHostFree(block);

    }
    GET_SYN_RES("controlRY")
}
bool GATEGPU:: controlRYdagger(QState& psigpu, Qnum& qnum, double theta, double error_rate)
{
    if (randGenerator() > error_rate)
    {
        double costheta = cos(theta / 2), sintheta = sin(theta / 2);
        QSIZE m = qnum.size();
        QSIZE target = qnum.back();
        sort(qnum.begin(), qnum.end());
        QSIZE *block, *blockgpu;
        hipHostAlloc((void **)&block, sizeof(QSIZE)*m, hipHostMallocMapped);
        hipHostGetDevicePointer(&blockgpu, block, 0);
        for (QSIZE i = 0; i < m; i++)
        {
            block[i] = 1 << qnum[i];
        }
        //QState* QPsi = (QState*)psi;
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        controlRYdagger << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), blockgpu, 1 << target, m, costheta, sintheta);
        hipHostFree(block);
    }
    GET_SYN_RES("controlRYdagger")
}
bool GATEGPU:: controlRZ(QState& psigpu, Qnum& qnum, double theta, double error_rate)
{
    if (randGenerator() > error_rate)
    {
        double costheta = cos(theta), sintheta = sin(theta);
        QSIZE m = qnum.size();
        QSIZE target = qnum.back();
        sort(qnum.begin(), qnum.end());
        QSIZE *block, *blockgpu;
        hipHostAlloc((void **)&block, sizeof(QSIZE)*m, hipHostMallocMapped);
        hipHostGetDevicePointer(&blockgpu, block, 0);
        for (QSIZE i = 0; i < m; i++)
        {
            block[i] = 1 << qnum[i];
        }
        //QState* QPsi = (QState*)psi;
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        controlRZ << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), blockgpu, 1 << target, m, costheta, sintheta);
        hipHostFree(block);

    }
    GET_SYN_RES("controlRZ")
}
bool GATEGPU:: controlRZdagger(QState& psigpu, Qnum& qnum, double theta, double error_rate)
{
    if (randGenerator() > error_rate)
    {
        double costheta = cos(theta), sintheta = sin(theta);
        QSIZE m = qnum.size();
        QSIZE target = qnum.back();
        sort(qnum.begin(), qnum.end());
        QSIZE *block, *blockgpu;
        hipHostAlloc((void **)&block, sizeof(QSIZE)*m, hipHostMallocMapped);
        hipHostGetDevicePointer(&blockgpu, block, 0);
        for (QSIZE i = 0; i < m; i++)
        {
            block[i] = 1 << qnum[i];
        }
        //QState* QPsi = (QState*)psi;
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        controlRZdagger << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), blockgpu, 1 << target, m, costheta, sintheta);
        hipHostFree(block);
    }
    GET_SYN_RES("controlRZdagger")
}
bool GATEGPU:: toffoli(QState& psigpu, size_t qn0, size_t qn1, size_t qn2, double error_rate)
{
    if (randGenerator() > error_rate)
    {
        //QState* QPsi = (QState*)psi;
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        toffoli << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), 1 << qn0, 1 << qn1, 1 << qn2);
    }
    GET_SYN_RES("toffoli")
}
bool GATEGPU:: toffolidagger(QState& psigpu, size_t qn0, size_t qn1, size_t qn2, double error_rate)
{
    if (randGenerator() > error_rate)
    {
        //QState* QPsi = (QState*)psi;
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        toffolidagger << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), 1 << qn0, 1 << qn1, 1 << qn2);
    }
    GET_SYN_RES("toffolidagger")
}
bool GATEGPU:: qbReset(QState& psigpu, size_t qn, double error_rate)
{
    if (randGenerator() > error_rate)
    {
        double * resultgpu;
        // hipHostAlloc((void **)&result, sizeof(STATE_T)*(psigpu.qnum-1))/THREADDIM, hipHostMallocMapped);
        //hipHostGetDevicePointer(&resultgpu, result, 0);
        hipMalloc((void **)&resultgpu, sizeof(STATE_T)*(1 << (psigpu.qnum - 1)) / THREADDIM);
        double * probgpu, *prob;
        hipHostAlloc((void **)&prob, sizeof(STATE_T), hipHostMallocMapped);
        hipHostGetDevicePointer(&probgpu, prob, 0);
        QSIZE BLOCKDIM;
        SET_BLOCKDIM
        qubitprob << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM, THREADDIM * sizeof(STATE_T) >> > (psigpu.real, psigpu.imag, 1 << (psigpu.qnum), 1 << qn, resultgpu);    //���ʵ�һ�ι�Լ
        probsum << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (resultgpu, probgpu);                   //Ҫ������̬�ĸ��ʴ���prob��
        hipDeviceSynchronize();           //�ȸ�����ȫ�������
        *prob = 1 / sqrt(*prob);
        qubitcollapse0 << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag, 1 << (psigpu.qnum), 1 << qn, *prob);
        hipDeviceSynchronize();           
        hipFree(resultgpu);
        hipHostFree(prob);
       // std::cout << "err = " << hipGetErrorString(hipDeviceSynchronize()) << endl;
    }
    GET_SYN_RES("qbReset")
    
}




/*
void GATEGPU:: unitarySingleQubitGate(double * psireal, double * psiimag, QSIZE Block, double *matr)
{
    double * matrgpu;
    hipMalloc((void GATEGPU:: **)(&matrgpu), sizeof(STATE_T) * 8);
    hipMemcpy((void *)matrgpu, (void *)matr, sizeof(STATE_T) * 8, hipMemcpyHostToDevice);
    unitary << < ((BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psireal, psiimag, matrgpu, Block);
    hipFree(&matrgpu);
}//checked
*/
int GATEGPU:: qubitmeasure(QState& psigpu, QSIZE Block)
{
    
    // double * result;
    double * resultgpu;
    // hipHostAlloc((void **)&result, sizeof(STATE_T)*(psigpu.qnum-1))/THREADDIM, hipHostMallocMapped);
    //hipHostGetDevicePointer(&resultgpu, result, 0);
    //QSIZE BLOCKDIM = (1 << (psigpu.qnum - 1)) / THREADDIM;
    QSIZE BLOCKDIM;
    SET_BLOCKDIM
    hipError_t cudaState = hipMalloc(&resultgpu, sizeof(STATE_T)* (BLOCKDIM == 0 ? 1 : BLOCKDIM));
    if (hipSuccess != cudaState)
    {
        cout << "resultgpu  " << hipGetErrorString(cudaState) << endl;
        return -1;
    }
    double * probgpu, prob;
    //hipHostAlloc((void **)&prob, sizeof(STATE_T), hipHostMallocMapped);
    //hipHostGetDevicePointer(&probgpu, prob, 0);
    cudaState = hipMalloc(&probgpu, sizeof(STATE_T));
    if (hipSuccess != cudaState)
    {
        cout << "probgpu  " << hipGetErrorString(cudaState) << endl;
        hipFree(resultgpu);
        return -1;
    }
    qubitprob << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM, THREADDIM * sizeof(STATE_T) >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), Block, resultgpu);    //���ʵ�һ�ι�Լ
    cudaState = hipDeviceSynchronize();           //�ȸ�����ȫ�������
    if (hipSuccess != cudaState)
    {
        cout << hipGetErrorString(cudaState) << endl;
        hipFree(resultgpu);
        hipFree(probgpu);
        return -1;
    }
    //double *prob;
    probsum << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (resultgpu, probgpu);                   //Ҫ������̬�ĸ��ʴ���prob��
    cudaState = hipDeviceSynchronize();           //�ȸ�����ȫ�������
    if (hipSuccess != cudaState)
    {
        cout << hipGetErrorString(cudaState) << endl;
        hipFree(resultgpu);
        hipFree(probgpu);
        return -1;
    }
    hipMemcpy(&prob, probgpu, sizeof(STATE_T), hipMemcpyDeviceToHost);
    cudaState = hipDeviceSynchronize();           //�ȸ�����ȫ�������
    if (hipSuccess != cudaState)
    {
        cout  << hipGetErrorString(cudaState) << endl;
        hipFree(resultgpu);
        hipFree(probgpu);
        return -1;
    }
                                       //hipMemcpy((void GATEGPU::*)&prob1, (void GATEGPU::*)prob, sizeof(STATE_T), hipMemcpyDeviceToHost);
    //dprob.prob = prob[0];
    //cout << prob[0] << "\t" << *prob << endl;
    //cout << "prob\t" << dprob.prob << endl;
    //*prob = prob1;
    int outcome = 0;
    if (randGenerator() > prob)
    {
        outcome = 1;
    }
    if (0 == outcome)
    {
       
        prob = 1 / sqrt(prob);
        qubitcollapse0 << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), Block, prob);
        //GET_SYN_RES("qubitmeasure")
    }
    else
    {
        
        prob = 1 / sqrt(1 - prob);
        qubitcollapse1 << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), Block, prob);
    //    GET_SYN_RES("qubitmeasure")
    }
    cudaState = hipFree(resultgpu);
    if (hipSuccess!=cudaState)
    {
        cout << "resultgpu free error" << endl;
        return -1;
    }
    cudaState =hipFree(probgpu);
    if (hipSuccess != cudaState)
    {
        cout << "probgpu free error" << endl;
        return -1;
    }
    //hipHostFree(prob);
    return outcome;
}//checked
 /*
 void GATEGPU:: Hadamard(QState& psi, size_t qn, double error_rate = 0)
 {

 Hadamard << < ((BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), 1 << qn);
 }*/

 //vecuint qnumtostate(vecuint& qn)
 //{

 //vecuint state(qn.size(), 0);

 //}
#if 0



bool GATEGPU:: pMeasure(QState& psigpu, vecprob& vprob,QSIZE * block, QSIZE m)
{
    QSIZE temp;
    for (int i = 0; i < m - 1; i++)
    {
        for (int j = 0; j < m - i - 1; j++)
        {
            if (block[j] > block[j + 1])
            {
                temp = block[j];
                block[j] = block[j + 1];
                block[j + 1] = temp;
            }
        }
    }//����
    double *probgpu;
    double * probc, *result;
    hipError_t cudaStatus=hipHostAlloc(&probc, sizeof(STATE_T)*(1 << m), hipHostMallocMapped);
    if (cudaStatus != hipSuccess)
    {
        printf("host alloc fail!\n");
        return true;
    }
    hipHostGetDevicePointer(&probgpu, probc, 0);
    //probc=(STATE_T*)malloc(sizeof(STATE_T)*(1<<m));          //����̬�ĸ���
    QSIZE *blockgpu1;                       //block
    QSIZE BLOCKDIM = (1 << (psigpu.qnum - 1)) / THREADDIM;
    hipMalloc((&blockgpu1), sizeof(QSIZE)*m);
    hipMalloc((&result), sizeof(STATE_T)*BLOCKDIM);          //����ʵ��м����
                                                                           //hipMalloc((void **)(&probgpu), sizeof(STATE_T)*(1<<m));
    hipMemcpy(blockgpu1, block, sizeof(QSIZE)*m, hipMemcpyHostToDevice);//blockgpu1 contains target qubit
    for (int i = 0; i < 1 << m; i++)
    {

        int index = 0;
        for (int j = 0; j < m; j++)
        {
            index += block[j] * ((i >> j) % 2);
        }
        //cout << "index\t" << index << endl;
        multiprob << < ((BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM, THREADDIM * sizeof(STATE_T) >> > (psigpu.real, psigpu.imag,1<<(psigpu.qnum), result, blockgpu1, m, index);
        //std::cout << "err1 = " << hipGetErrorString(hipDeviceSynchronize()) << endl;
        probsum << < ((BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (result, &probgpu[i]);
        //std::cout << "err2 = " << hipGetErrorString(hipDeviceSynchronize()) << endl;
        //hipMemcpy((void GATEGPU::*)probc, (void GATEGPU::*)probgpu, sizeof(STATE_T)*(1<<m), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        //vprob[i].first = i;
        //vprob[i].second = probc[i];
        vprob.push_back(GPUPAIR(i, probc[i]));

    }
    //std::cout << *probc << endl;
    hipFree(result);
    hipFree(blockgpu1);
    hipHostFree(probc);
    return ;
}
#endif // 0
#if 1

bool probcompare(pair<size_t, double>& a, pair<size_t, double>& b)
{
    return a.second> b.second;
}

bool GATEGPU::pMeasurenew(QState& psigpu, vector<pair<size_t, double>>& vprob, Qnum& qnum)
{
    hipDeviceSynchronize();
    QSIZE m = qnum.size();
    sort(qnum.begin(),qnum.end());
    if (m <= psigpu.qnum / 2)
    {
        QSIZE *block, *blockgpu;
        hipHostAlloc(&block, sizeof(QSIZE)*m, hipHostMallocMapped);
        hipHostGetDevicePointer(&blockgpu, block, 0);
        QSIZE temp;
        for (size_t i = 0; i < m; i++)
        {
            block[i] = 1 << qnum[i];
        }//����
        double *probgpu;
        double * probc, *result;
        hipError_t cudaStatus = hipHostAlloc(&probc, sizeof(STATE_T)*(1 << m), hipHostMallocMapped);
        if (cudaStatus != hipSuccess)
        {
            printf("host alloc fail!\n");
            return false;
        }
        hipHostGetDevicePointer(&probgpu, probc, 0);
        //probc=(STATE_T*)malloc(sizeof(STATE_T)*(1<<m));          //����̬�ĸ���
        QSIZE *blockgpu1;                       //block
        QSIZE BLOCKDIM = (1u << (psigpu.qnum - 1)) / THREADDIM;
        hipMalloc((&blockgpu1), sizeof(QSIZE)*m);
        hipMalloc((&result), sizeof(double)*(BLOCKDIM == 0 ? 1 : BLOCKDIM));          //����ʵ��м����
                                                                                       //hipMalloc((void **)(&probgpu), sizeof(STATE_T)*(1<<m));
        hipMemcpy(blockgpu1, block, sizeof(QSIZE)*m, hipMemcpyHostToDevice);
        for (size_t i = 0; i < 1u << m; i++)
        {
            size_t index = 0;
            for (size_t j = 0; j < m; j++)
            {
                index += block[j] * ((i >> j) % 2);
            }
            // cout << "index\t" << index << endl;
            multiprob << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM, THREADDIM * sizeof(STATE_T) >> > (psigpu.real, psigpu.imag, 1 << (psigpu.qnum), result, blockgpu1, m, index);
            probsum << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (result, probgpu + i);
            //hipMemcpy((void GATEGPU::*)probc, (void GATEGPU::*)probgpu, sizeof(STATE_T)*(1<<m), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            //vprob[i].first = i;
            //vprob[i].second = probc[i];
            vprob.push_back(GPUPAIR(i, probc[i]));

        }
        if (vprob.size() <= 10)
        {
            sort(vprob.begin(), vprob.end(), probcompare);
        }
        else
        {
            sort(vprob.begin(), vprob.end(), probcompare);
            vprob.erase(vprob.begin() + 10, vprob.end());
        }
        //std::cout << *probc << endl;
        hipFree(result);
        hipFree(blockgpu1);
        hipHostFree(probc);
        hipHostFree(block);
        return true;
    }
    else
    {
        size_t Dim = 1u << psigpu.qnum;
        size_t blocknum = 1u << (m - psigpu.qnum / 4);         //blocknum��ʾblock��
        STATE_T *probtemp, *probtempgpu;
        hipError_t cudastate;
        cudastate = hipHostAlloc(&probtemp, sizeof(double) * blocknum, hipHostMallocMapped);
        if (cudastate != hipSuccess)
        {
            hipHostFree(probtemp);
            return false;
        }
        cudastate = hipHostGetDevicePointer(&probtempgpu, probtemp, 0);
        if (cudastate != hipSuccess)
        {
            hipHostFree(probtemp);
            return false;
        }
        size_t *block, *blockgpu;
        cudastate = hipHostAlloc(&block, sizeof(size_t)*m, hipHostMallocMapped);
        if (cudastate != hipSuccess)
        {
            hipHostFree(probtemp);
            hipHostFree(block);
            return false;
        }
        cudastate = hipHostGetDevicePointer(&blockgpu, block, 0);
        if (cudastate != hipSuccess)
        {
            hipHostFree(probtemp);
            hipHostFree(block);
            return false;
        }
        for (size_t i = 0; i < m; i++)
        {
            block[i] = 1u << qnum[i];
        }//����
        for (size_t i = 0; i < blocknum; i++)
        {
            probtemp[i] = 0;
        }
        for (size_t i = 0; i < 10; i++)
        {
            vprob.push_back(GPUPAIR(0, 0));
        }
        size_t BLOCKDIM = blocknum / THREADDIM;
        for (size_t i = 0; i < (1u << m); i += blocknum)
        {
            //(STATE_T * psireal, STATE_T * psiimag, STATE_T *probtemp, size_t num1, size_t m, size_t Dim, size_t * block)
            
            probsumnew1 << < (BLOCKDIM == 0 ? 1 : BLOCKDIM), THREADDIM >> > (psigpu.real, psigpu.imag, probtempgpu, i, m, Dim, blockgpu);
            
            cudastate =  hipDeviceSynchronize();
            if (hipSuccess!=cudastate)
            {
                cout << "error" << endl;
            }
            for (size_t j = 0; j < blocknum; j++)
            {
                if (probtemp[j] > vprob[9].second)
                {
                    vprob[9] = GPUPAIR(i + j, probtemp[j]);
                    sort(vprob.begin(), vprob.end(), probcompare);
                }
            }
        }
        hipHostFree(probtemp);
        hipHostFree(block);
        return true;
    }

}

#endif // 0


#if 0
bool GATEGPU::pMeasurenew(QState& psigpu, vector<pair<int, double>>& vprob, Qnum& qnum)
{
    QSIZE dim = (1 << psigpu.qnum);

    double* psiHostReal = (double*)malloc(dim * sizeof(double));

    double* psiHostImag = (double*)malloc(dim * sizeof(double));

    hipMemcpy(psiHostReal, psigpu.real, dim * sizeof(double), hipMemcpyDeviceToHost);

    hipMemcpy(psiHostImag, psigpu.imag, dim * sizeof(double), hipMemcpyDeviceToHost);

    vprob.resize((1 << qnum.size()));

    for (int i = 0; i < (1 << qnum.size()); i++)
    {
        vprob[i].first = i;
        vprob[i].second = 0;
    }

    for (int i = 0; i <dim; i++)
    {
        volatile int idx = 0;
        for (int j = 0; j < qnum.size(); j++)
        {
            idx += (((i >> (qnum[j])) & 1) << (qnum.size() - 1 - j));
        }
        vprob[idx].second +=
            (
                abs(psiHostReal[i])*abs(psiHostReal[i])
                +
                abs(psiHostImag[i])*abs(psiHostImag[i])
                );
    }

    if (vprob.size() <= 10)
    {
        sort(vprob.begin(), vprob.end(), probcompare);
    }
    else
    {
        sort(vprob.begin(), vprob.end(), probcompare);
        vprob.erase(vprob.begin() + 10, vprob.end());
    }

    free(psiHostReal);
    free(psiHostImag);
    return true;
}
#endif  //0
bool GATEGPU::getState(QState &psi,QState &psigpu,int qnum)
{
	if (qnum < 30)
    {
        QSIZE Dim = 1 << qnum;
        hipMemcpy(psi.real, psigpu.real, sizeof(STATE_T)*Dim, hipMemcpyDeviceToHost);
        hipMemcpy(psi.imag, psigpu.imag, sizeof(STATE_T)*Dim, hipMemcpyDeviceToHost);
    }
    return true;
}

double GATEGPU:: randGenerator()
{
    int  ia = 16807, im = 2147483647, iq = 127773, ir = 2836;           /*difine constant number in 16807 generator.*/
    time_t rawtime;
    struct tm * timeinfo;
    time(&rawtime);
    timeinfo = localtime(&rawtime);
    static int irandseed = timeinfo->tm_year + 70 *
        (timeinfo->tm_mon + 1 + 12 *
        (timeinfo->tm_mday + 31 *
            (timeinfo->tm_hour + 23 *
            (timeinfo->tm_min + 59 * timeinfo->tm_sec))));
    static int irandnewseed;
    if (ia*(irandseed%iq) - ir*(irandseed / iq) >= 0)
        irandnewseed = ia*(irandseed%iq) - ir*(irandseed / iq);
    else
        irandnewseed = ia*(irandseed%iq) - ir*(irandseed / iq) + im;
    irandseed = irandnewseed;
    return (double)irandnewseed / im;

}
